// Niko Galedo
// CSC 656
// Vector Addition on GPU instead of CPU


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


// this code came from the CUDA C Programming Guide

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
 for (int i = 0; i < n; i++)
   y[i] = x[i] + y[i];
}
 
int main(void)
{
 // step 3 says to change the size of the arrays to 512M elements
 int N = 1<<29; // 512M  elements
 float *x, *y;
 
// Allocate Unified Memory – accessible from CPU or GPU
 hipMallocManaged(&x, N*sizeof(float));
 hipMallocManaged(&y, N*sizeof(float));
 
 // initialize x and y arrays on the host
 for (int i = 0; i < N; i++) {
   x[i] = 1.0f;
   y[i] = 2.0f;
 }
 
 // Launch kernel with 1 thread
 add<<<1, 1>>>(N, x, y);

 // Wait for GPU to finish before accessing on host
 hipDeviceSynchronize();

 // Check for errors (all values should be 3.0f)
 float maxError = 0.0f;
 for (int i = 0; i < N; i++)
   maxError = fmax(maxError, fabs(y[i]-3.0f));
 std::cout << "Max error: " << maxError << std::endl;
 
 // Free memory
 hipFree(x);
 hipFree(y);
 
 return 0;
}